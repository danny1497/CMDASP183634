#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"

// This is my DEVICE function
// __global__ means this function is visible to the host
__global__ void kernelHelloWorld() {

	int thread = threadIdx.x; //local thread number in a block
	int block =  blockIdx.x; //block nunmber 
		
	printf("Hello World %d of block %d!\n", thread, block);
	
}

int main(int argc, char** argv) {

	int Nblocks = 10; // Number of blocks 
	int Nthreads = 3; // Number of threads

	// run the function 'kernelHelloWorld' on the DEVICE
	kernelHelloWorld <<< Nblocks, Nthreads >>> ();

	// wait for the DEVICE function to complete before moving on
	hipDeviceSynchronize();
	
	return 0;
		
}
