#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"

__device__ int cudamodprod(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int za = a;
  unsigned int ab = 0;

  while (b > 0) {
    if (b%2 == 1) ab = (ab +  za) % p;
    za = (2 * za) % p;
    b /= 2;
  }
  return ab;
}

//compute a^b mod p safely
__device__ int cudamodExp(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int z = a;
  unsigned int aExpb = 1;

  while (b > 0) {
    if (b%2 == 1) aExpb = cudamodprod(aExpb, z, p);
    z = cudamodprod(z, z, p);
    b /= 2;
  }
  return aExpb;
}

__global__ void findSecretKey(unsigned int g, unsigned int h, unsigned int p, unsigned int *dx) {

  int threadID = threadIdx.x;
  int blockID = blockIdx.x;
  int Nblock = blockDim.x;
  unsigned int id = threadID + (blockID*Nblock);
  if(id<(p-1)) {
    if (cudamodExp(g,id+1,p)==h) {
      *dx=id+1;
    }
  }
}

int main (int argc, char **argv) {

  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */

  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */
  
  //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;
  int Nthreads = atoi(argv[1]);
  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */
  //FILE *pubKey,*msg;
  unsigned int Nblocks;
  //unsigned int Nthreads,blocks;
  FILE *pubKey = fopen("public_key.txt","r");
  FILE *msg = fopen("message.txt","r");
  fscanf(pubKey, "%u\n%u\n%u\n%u\n",&n,&p,&g,&h);
  //printf("n= %u, p = %u, g = %u, h = %u\n",n,p,g,h);
  fscanf(msg, "%u\n", &Nints);
  //printf("Nints  = %u\n", Nints);
  unsigned int charsPerInt = (n-1)/8;
  unsigned int *Zmessage = (unsigned int *) malloc(Nints*sizeof(unsigned int));
  unsigned int *a = (unsigned int *) malloc(Nints*sizeof(unsigned int));

  for (unsigned int i = 0; i < Nints; i++) {
    fscanf(msg, "%u %u\n", &Zmessage[i],&a[i]);
    //printf("Zmessage = %u, a = %u\n",Zmessage[i],a[i]);
  }

  fclose(pubKey);
  fclose(msg);
  unsigned int host = 0;
  unsigned int *device;
  hipMalloc(&device,sizeof(unsigned int));
  // find the secret key
  /*
  if (x==0 || cudamodExp(g,x,p)!=h) {
    printf("Finding the secret key...\n");
    double startTime = clock();
    for (unsigned int i=0;i<p-1;i++) {
      if (cudamodExp(g,i+1,p)==h) {
        printf("Secret key found! x = %u \n", i+1);
        x=i+1;
      } 
    }*/
  if (x==0 || cudamodExp(g,x,p)!=h) {
    printf("Finding the secret key...\n");
  }
  double startTime = clock();   
  //Nthreads = 32;
  blocks = (p-1)/32;
  findSecretKey <<<Nblocks,32>>>(g,h,p,device);
  hipDeviceSynchronize();
  unsigned int size = sizeof(unsigned int);
  hipMemcpy(host,device,size, cudeMemcpyDeviceToHost);
  double endTime = clock();

  double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
  double work = (double) p;
  double throughput = work/totalTime;
  printf("Secret key found! x %u \n", host);
  printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
  //}

  //int buffer = 1024;
  //ElGamalDecrypt(Zmessage,a,Nints,p,x);
  unsigned int Nchars = Nints*charsPerInt;
  unsigned char *message = (unsigned char *) malloc(buffer*sizeof(unsigned char));
  ElGamalDecrypt(Zmessage,a,Nints,p,x);
  //unsigned int cpi = (n-1)/8;
  convertZToString(Zmessage,Nints,message,Nchars);
  printf("Decrypted message: \"%s\"\n",message);
  hipFree(device); 
  return 0;
}
