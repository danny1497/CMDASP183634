#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"

__device__ int cudamodprod(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int za = a;
  unsigned int ab = 0;

  while (b > 0) {
    if (b%2 == 1) ab = (ab +  za) % p;
    za = (2 * za) % p;
    b /= 2;
  }
  return ab;
}

//compute a^b mod p safely
__device__ int cudamodExp(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int z = a;
  unsigned int aExpb = 1;

  while (b > 0) {
    if (b%2 == 1) aExpb = cudamodprod(aExpb, z, p);
    z = cudamodprod(z, z, p);
    b /= 2;
  }
  return aExpb;
}

__global__ void findSecretKey(unsigned int *dx, unsigned int p, unsigned int g, unsigned int h) {

  int threadID = threadIdx.x;
  int blockID = blockIdx.x;
  int Nblock = blockDim.x;
  unsigned int id = threadID + (blockID*Nblock);
  //if(id<(*p-1)) {
    if (cudamodExp(g,id,p)==h) {
      dx[0]=id;
    }
  //}
}

int main (int argc, char **argv) {

  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */

  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */
  
  //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;
  int Nthreads = atoi(argv[1]);
  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */
  //FILE *pubKey,*msg;
  //unsigned int Nblocks;
  //unsigned int Nthreads,blocks;
  //FILE *pubKey = fopen("public_key.txt","r");
  //FILE *msg = fopen("message.txt","r");
	FILE *pubKey = fopen("bonus_public_key.txt","r");
  	FILE *msg = fopen("bonus_message.txt","r");
	
  fscanf(pubKey, "%u\n%u\n%u\n%u\n",&n,&p,&g,&h);
  //printf("n= %u, p = %u, g = %u, h = %u\n",n,p,g,h);
  fscanf(msg, "%u\n", &Nints);
  //printf("Nints  = %u\n", Nints);
  //unsigned int charsPerInt = (n-1)/8;
  unsigned int *Zmessage = (unsigned int *) malloc(Nints*sizeof(unsigned int));
  unsigned int *a = (unsigned int *) malloc(Nints*sizeof(unsigned int));

  for (unsigned int i = 0; i < Nints; i++) {
    fscanf(msg, "%u %u\n", &Zmessage[i],&a[i]);
    //printf("Zmessage = %u, a = %u\n",Zmessage[i],a[i]);
  }

  fclose(pubKey);
  fclose(msg);
  //unsigned int host = 0;
  //unsigned int *dx,*dp,*dg,*dh;
  /*hipMalloc(&dx,sizeof(unsigned int));
	hipMalloc(&dg,sizeof(unsigned int));
	hipMalloc(&dh,sizeof(unsigned int));
	hipMalloc(&dp,sizeof(unsigned int));
 	hipMemcpy(dp,&g,sizeof(unsigned int),hipMemcpyHostToDevice);
	hipMemcpy(dg,&h,sizeof(unsigned int),hipMemcpyHostToDevice);
	hipMemcpy(dh,&p,sizeof(unsigned int),hipMemcpyHostToDevice);
	*/

	// find the secret key
  /*
  if (x==0 || cudamodExp(g,x,p)!=h) {
    printf("Finding the secret key...\n");
    double startTime = clock();
    for (unsigned int i=0;i<p-1;i++) {
      if (cudamodExp(g,i+1,p)==h) {
        printf("Secret key found! x = %u \n", i+1);
        x=i+1;
      } 
    }*/
	double startTime = clock();
	//int thds = 1024;	
	//int Nblocks = (p-1+thds-1)/thds;
	unsigned int *host, *device;
	host = (unsigned int*) malloc(sizeof(unsigned int));
	hipMalloc(&device,sizeof(unsigned int));
	int xloc = 1;
	int yloc = 1;
	int zloc = 1;	
	dim3 Block(Nthreads,yloc,zloc);
	dim3 Grid((p+Nthreads-xloc)/Nthreads,yloc,zloc);
	findSecretKey <<<Grid,Block>>>(device,p,g,h);
	hipDeviceSynchronize();
	double endTime = clock();

  //if (x==0 || modExp(g,x,p)!=h) {
    //printf("Finding the secret key...\n");
  
  //double startTime = clock();   
  //Nthreads = 32;
  //Nblocks = (p-1)/32;
  //findSecretKey <<<Nblocks,thds>>>(dg,dh,dp,dx);
  //hipDeviceSynchronize();
  //unsigned int size = sizeof(unsigned int);
  hipMemcpy(host,device,sizeof(unsigned int), hipMemcpyDeviceToHost);
  //double endTime = clock();
	x = host[0];
	ElGamalDecrypt(Zmessage,a,Nints,p,x);
	unsigned int charsPerInt = (n-1)/8;
	int buffer = 1024;
	unsigned int Nchars = Nints*charsPerInt;
	unsigned char *result = (unsigned char *) malloc(buffer*sizeof(unsigned char));
	convertZToString(Zmessage,Nints, result, Nchars);
	printf("Decrypted message: \"%s\"\n",result);
	//printf("\n");
  double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
  double work = (double) p;
  double throughput = work/totalTime;
  //printf("Secret key found! x %u \n", host);
  printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
	//}
  /*
	hipFree(dg);
	hipFree(dh);
	hipFree(dp);
	hipFree(dx);
	//}
	*/
	hipFree(device);
	free(host);
	//free(Zmessage);
	//free(a);
	//free(result);
  //int buffer = 1024;
  //ElGamalDecrypt(Zmessage,a,Nints,p,x);
  //unsigned int Nchars = Nints*charsPerInt;
	/*int buffer = 1024;
	unsigned int temp = (n-1)/8;
  //unsigned int charsPerInt = temp/Nints;
	unsigned char *result = (unsigned char *) malloc(buffer*sizeof(unsigned char));
  ElGamalDecrypt(Zmessage,a,Nints,p,x);
  //unsigned int cpi = (n-1)/8;
  convertZToString(Zmessage,Nints,result,Nints*temp);
  printf("Decrypted message: \"%s\"\n",result);
  //hipFree(device); 
  */return 0;
}
