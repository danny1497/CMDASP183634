#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"


int main (int argc, char **argv) {

  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */

  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */
  //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */
  //FILE *pubKey,*msg;
  FILE *pubKey = fopen("public_key.txt","r");
  FILE *msg = fopen("message.txt","r");
  fscanf(pubKey, "%u\n%u\n%u\n%u\n",&n,&p,&g,&h);
  //printf("n= %u, p = %u, g = %u, h = %u\n",n,p,g,h);
  fscanf(msg, "%u\n", &Nints);
  //printf("Nints  = %u\n", Nints);
  unsigned int charsPerInt = (n-1)/8;
  unsigned int *Zmessage = (unsigned int *) malloc(Nints*sizeof(unsigned int));
  unsigned int *a = (unsigned int *) malloc(Nints*sizeof(unsigned int));

  for (unsigned int i = 0; i < Nints; i++) {
    fscanf(msg, "%u %u\n", &Zmessage[i],&a[i]);
    //printf("Zmessage = %u, a = %u\n",Zmessage[i],a[i]);
  }

  fclose(pubKey);
  fclose(msg);

  // find the secret key
  if (x==0 || modExp(g,x,p)!=h) {
    printf("Finding the secret key...\n");
    double startTime = clock();
    for (unsigned int i=0;i<p-1;i++) {
      if (modExp(g,i+1,p)==h) {
        printf("Secret key found! x = %u \n", i+1);
        x=i+1;
      } 
    }
    double endTime = clock();

    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;

    printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
  }
  return 0;
}
